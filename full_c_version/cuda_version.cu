#include "hip/hip_runtime.h"
#include <stdexcept>
#include <vector>
#include <iostream>

#include <hip/hip_runtime.h>
#include <stdint.h>

#include "cudaMallocWrapper.hpp"
#include "dbg.h"


#define FUNCS_PER_KERNEL 8192



// Some static variables for the CUDA version. That's not beautiful, I know.
struct GPUProps {
    bool initialized;
    int maxThreadsPerBlock;
    int maxConcurrentBlocks;
    int maxConcurrentThreads;
};

static struct GPUProps GPUProps = {false, 0, 0, 0};

void getGPUProperties()
{
    if (GPUProps.initialized) {
        return;
    }

    hipError_t ret;
    struct hipDeviceProp_t deviceProp;

    ret = hipGetDeviceProperties(&deviceProp, 0);
    if (ret != hipSuccess) {
        throw std::runtime_error("Failed to get device properties: " + std::string(hipGetErrorString(ret)));
    }

    GPUProps.maxThreadsPerBlock = deviceProp.maxThreadsDim[0];
    GPUProps.maxConcurrentBlocks = deviceProp.maxGridSize[0];
    GPUProps.maxConcurrentThreads = GPUProps.maxThreadsPerBlock * GPUProps.maxConcurrentBlocks;

    GPUProps.initialized = true;
}






/**********************************************************************************************
 ********************************* Form x0 + a + b + cd ***************************************
 **********************************************************************************************/


struct Function_0_a_b_cd {
    uint8_t a;
    uint8_t b;
    uint8_t c;
    uint8_t d;
    uint8_t nVariables;
};


bool smaller_or_equal_0_a_b_cd(struct Function_0_a_b_cd& one, struct Function_0_a_b_cd& other)
{
    if (one.a < other.a)
        return true;

    if (one.a == other.a && one.b < other.b)
        return true;

    if (one.a == other.a && one.b == other.b && one.c < other.c)
        return true;

    if (one.a == other.a && one.b == other.b && one.c == other.c && one.d <= other.d)
        return true;

    return false;
}

bool canonical_0_a_b_cd(struct Function_0_a_b_cd& func)
{
    int32_t ar = func.nVariables - func.a;
    int32_t br = func.nVariables - func.b;
    int32_t cr = func.nVariables - func.c;
    int32_t dr = func.nVariables - func.d;

    struct Function_0_a_b_cd other = {br, ar, dr, cr, func.nVariables};
    return smaller_or_equal_0_a_b_cd(func, other);
}


#define bit(nBit, val) (((val) >> (nBit)) & 1)


__global__ void kernel_0_a_b_cd(struct Function_0_a_b_cd *d_funcArray, uint32_t *d_funcLength,
                                uint32_t nQueued, uint32_t maxPossibleLength)
{
    // Get my position
    uint32_t me = blockIdx.x * blockDim.x + threadIdx.x;

    if (me < nQueued) {
        // Copy things into registers
        uint8_t a = d_funcArray[me].a;
        uint8_t b = d_funcArray[me].b;
        uint8_t c = d_funcArray[me].c;
        uint8_t d = d_funcArray[me].d;
        uint8_t nVariables = d_funcArray[me].nVariables;

        uint32_t curVal = 1;
        uint32_t length = 0;
        uint32_t newBit = 0;

        do {
            newBit = bit(0, curVal) ^ bit(a, curVal) ^ bit(b, curVal) ^
                    (bit(c, curVal) & bit(d, curVal));
            curVal = (curVal >> 1) | (newBit << (nVariables - 1));

            length++;
        } while (curVal != 1);

        d_funcLength[me] = length;
    }
}







void sendAndReport_0_a_b_cd(struct Function_0_a_b_cd *h_funcArray, uint32_t *h_funcLength,
                            struct Function_0_a_b_cd *d_funcArray, uint32_t *d_funcLength,
                            uint32_t maxPossibleLength, uint32_t nQueued)
{
    uint32_t nBlocks, nThreadsPerBlock;

    /*log_info("Sending %u functions to device\n", nQueued);*/

    cudaMemcpyWrapped<Function_0_a_b_cd>(d_funcArray, h_funcArray, nQueued, hipMemcpyHostToDevice);

    nBlocks = nQueued / GPUProps.maxThreadsPerBlock + (nQueued % GPUProps.maxThreadsPerBlock == 0 ? 0 : 1);

    nThreadsPerBlock = (nBlocks == 1) ? nQueued : GPUProps.maxThreadsPerBlock;

    /*log_info("Launching kernel");*/

    kernel_0_a_b_cd <<< nBlocks, nThreadsPerBlock >>> (d_funcArray, d_funcLength, nQueued, maxPossibleLength);

    cudaMemcpyWrapped<uint32_t>(h_funcLength, d_funcLength, nQueued, hipMemcpyDeviceToHost);

    /*log_info("Kernel returned");*/

    for (int i = 0; i < nQueued; i++) {
        std::cout << h_funcLength[i] << std::endl;
#if 0
        if (h_funcLength[i] == maxPossibleLength) {
            std::cout << "0," << (uint32_t)h_funcArray[i].a << "," << (uint32_t)h_funcArray[i].b
                      << ",(" << (uint32_t)h_funcArray[i].c << "," << (uint32_t)h_funcArray[i].d << ")"
                      << std::endl;
        }
#endif
    }

    std::cout << maxPossibleLength << std::endl;
}




/************************************ Functions generation **************************************/
void report_0_a_b_cd(uint32_t nVariables)
{
    getGPUProperties();

    //XXX Carefull with nVariables 32.
    uint32_t maxPossibleLength = (1 << nVariables) - 1;

    std::vector<Function_0_a_b_cd> h_funcArray;
    std::vector<uint32_t> h_funcLength(FUNCS_PER_KERNEL);

    /* Allocate memory for arrays on device */
    CudaMallocWrapper<Function_0_a_b_cd> d_funcArray(FUNCS_PER_KERNEL);
    CudaMallocWrapper<uint32_t> d_funcLength(FUNCS_PER_KERNEL);


    /* Generate the functions */
    for (int32_t a = 1; a <= (nVariables + 1) / 2; a++) {
        for (int32_t b = a + 1; b <= nVariables - 1; b++) {

            for (int32_t c = 1; c <= nVariables - 2; c++) {
                for (int32_t d = c + 1; d <= nVariables - 1; d++) {

                    // Keep the function for later evaluation
                    struct Function_0_a_b_cd func = {a, b, c, d, nVariables};
                    if (!canonical_0_a_b_cd(func)) {
                        continue;
                    }

                    h_funcArray.push_back(func);

                    if (h_funcArray.size() == FUNCS_PER_KERNEL) {
                        // Warning with nvcc for taking adress of a stack variable, but no problem here
                        sendAndReport_0_a_b_cd((struct Function_0_a_b_cd*)&h_funcArray[0], (uint32_t*)&h_funcLength[0],
                                               (struct Function_0_a_b_cd*)d_funcArray.mem, (uint32_t*)d_funcLength.mem,
                                                maxPossibleLength, h_funcArray.size());

                        h_funcArray.clear();
                    }
                }
            }
        }
    }

    if (h_funcArray.size() != 0) {
        sendAndReport_0_a_b_cd((struct Function_0_a_b_cd*)&h_funcArray[0], (uint32_t *)&h_funcLength[0],
                               (struct Function_0_a_b_cd*)d_funcArray.mem, (uint32_t *)d_funcLength.mem,
                                maxPossibleLength, h_funcArray.size());
    }
}







#if 0






    template <class FunctionType>
__global__ void kernel(FunctionType *d_funcArray, bool *d_isMaxLength, uint32_t nFunctions, uint32_t maxPossibleLength)
{
    // Get my position in the grid
    uint32_t me = blockIdx.x * blockDim.x + threadIdx.x;

    if (me < nFunctions) {
        if (d_funcArray[me].getCycleLength_device() == maxPossibleLength) {
            d_isMaxLength[me] = true;
        } else {
            d_isMaxLength[me] = false;
        }
    }

}







    template <class FunctionType>
static void sendAndReport(FunctionType *h_funcArray, FunctionType *d_funcArray,
        bool *h_isMaxLength, bool *d_isMaxLength, uint32_t enqueued, uint32_t maxPossibleLength)
{
    hipError_t ret;
    uint32_t nBlocks, nThreadsPerBlock;

    std::cerr << "Sending " << enqueued << " functions to device" << std::endl;

    ret = hipMemcpy(d_funcArray, h_funcArray, enqueued * sizeof(FunctionType), hipMemcpyHostToDevice);
    if (ret != hipSuccess) {
        throw std::runtime_error("Failed to memcpy to device: " + std::string(hipGetErrorString(ret)));
    }

    nBlocks = enqueued / GPUProps.maxThreadsPerBlock + (enqueued % GPUProps.maxThreadsPerBlock == 0 ? 0 : 1);

    nThreadsPerBlock = (nBlocks == 1) ? enqueued : GPUProps.maxThreadsPerBlock;

    std::cerr << "Launching kernel" << std::endl;
    // Launch kernel
    kernel<FunctionType> <<< nBlocks, nThreadsPerBlock >>> (d_funcArray, d_isMaxLength, enqueued, maxPossibleLength);

    ret = hipMemcpy(h_isMaxLength, d_isMaxLength, enqueued * sizeof(bool), hipMemcpyDeviceToHost);
    if (ret != hipSuccess) {
        throw std::runtime_error("Kernel execution failed: " + std::string(hipGetErrorString(ret)));
    }

    std::cerr << "Kernel has returned successfully" << std::endl;

    for (int i = 0; i < enqueued; i++) {
        if (h_isMaxLength[i]) {
            std::cout << h_funcArray[i].toString() << std::endl;
        }
    }
}




/***********************************************************************
 ********************* For x0 + xa + xb + xc.xd ************************
 ***********************************************************************/

    FuncGenerator_0_a_b_cd::FuncGenerator_0_a_b_cd(uint32_t nVariables)
: m_nVariables(nVariables), m_maxPossibleLength((1 << nVariables) - 1)
{}

FuncGenerator_0_a_b_cd::~FuncGenerator_0_a_b_cd() {}







/***********************************************************************
 ******************** For x0 + xa + xb.xc + xd.xe **********************
 ***********************************************************************/

    FuncGenerator_0_a_bc_de::FuncGenerator_0_a_bc_de(uint32_t nVariables)
: m_nVariables(nVariables), m_maxPossibleLength((1 << m_nVariables) - 1)
{}

FuncGenerator_0_a_bc_de::~FuncGenerator_0_a_bc_de() {}




void FuncGenerator_0_a_bc_de::reportMaxFunctions()
{
    getGPUProperties();

    // The lexicographical order is difficult to handle in the generation
    // for b,c and d,e. So this will be handled in isCanonicalForm()

    // d can start from b, because if d < b, then a commutatively equivalent function
    // will have been tested (as b.c and d.e can commute around +), and that variant
    // would be smaller by lexicographical order.

    // We don't want b = c AND d = e either, which gives us kind of a "degenerated" function.
    // This is also handled in isCanonicalForm()

    hipError_t ret;

    std::vector<Function_0_a_bc_de> h_funcVector;
    Function_0_a_bc_de *h_funcArray;
    Function_0_a_bc_de *d_funcArray;
    bool *d_isMaxLength;
    bool *h_isMaxLength;

    h_isMaxLength = new bool[GPUProps.actualConcurrentThreads];
    h_funcArray = new Function_0_a_bc_de[GPUProps.actualConcurrentThreads];

    ret = hipMalloc((void **) &d_isMaxLength, GPUProps.actualConcurrentThreads * sizeof(bool));
    if (ret != hipSuccess) {
        throw std::runtime_error("No more memory on device for bools");
    }

    ret = hipMalloc((void **) &d_funcArray, GPUProps.actualConcurrentThreads * sizeof(Function_0_a_bc_de));
    if (ret != hipSuccess) {
        throw std::runtime_error("No more memory on device for Function_0_a_bc_de");
    }

    uint32_t enqueued = 0;

    for (int32_t a = 1; a <= (m_nVariables + 1) / 2; a++) {

        for (int32_t b = 1; b <= m_nVariables - 2; b++) {
            for (int32_t c = b + 1; c <= m_nVariables - 1; c++) {

                for (int32_t d = b; d <= m_nVariables - 2; d++) {
                    for (int32_t e = d + 1; e <= m_nVariables - 1; e++) {

                        // Keep the function for later evaluation
                        Function_0_a_bc_de func(a, b, c, d, e, m_nVariables);
                        if (!func.isCanonicalForm()) {
                            continue;
                        }

                        h_funcArray[enqueued++] = func;

                        if (enqueued == GPUProps.actualConcurrentThreads) {
                            sendAndReport<Function_0_a_bc_de>(h_funcArray, d_funcArray, h_isMaxLength,
                                    d_isMaxLength, enqueued, m_maxPossibleLength);
                            enqueued = 0;
                        }
                    }
                }
            }
        }
    }

    if (enqueued != 0) {
        sendAndReport<Function_0_a_bc_de>(h_funcArray, d_funcArray, h_isMaxLength,
                d_isMaxLength, enqueued, m_maxPossibleLength);
    }

    hipFree(d_funcArray);
    hipFree(d_isMaxLength);
    delete [] h_isMaxLength;
    delete [] h_funcArray;
}




/***********************************************************************
 **************** For x0 + xa + xb + xc + xd + xe.xf *******************
 ***********************************************************************/

    FuncGenerator_0_a_b_c_d_ef::FuncGenerator_0_a_b_c_d_ef(uint32_t nVariables)
: m_nVariables(nVariables), m_maxPossibleLength((1 << m_nVariables) - 1)
{}


FuncGenerator_0_a_b_c_d_ef::~FuncGenerator_0_a_b_c_d_ef()
{}


void FuncGenerator_0_a_b_c_d_ef::reportMaxFunctions()
{
    getGPUProperties();

    hipError_t ret;

    Function_0_a_b_c_d_ef *h_funcArray;
    Function_0_a_b_c_d_ef *d_funcArray;
    bool *d_isMaxLength;
    bool *h_isMaxLength;

    h_isMaxLength = new bool[GPUProps.actualConcurrentThreads];
    h_funcArray = new Function_0_a_b_c_d_ef[GPUProps.actualConcurrentThreads];

    ret = hipMalloc((void **) &d_isMaxLength, GPUProps.actualConcurrentThreads * sizeof(bool));
    if (ret != hipSuccess) {
        throw std::runtime_error("No more memory on device for bools");
    }

    ret = hipMalloc((void **) &d_funcArray, GPUProps.actualConcurrentThreads * sizeof(Function_0_a_b_c_d_ef));
    if (ret != hipSuccess) {
        throw std::runtime_error("No more memory on device for Function_0_a_b_c_d_ef");
    }

    uint32_t enqueued = 0;

    for (int32_t a = 1; a <= (m_nVariables + 1) / 2; a++) {
        for (int32_t b = a + 1; b <= m_nVariables - 3; b++) { /* -3 to leave room for c and d */
            for (int32_t c = b + 1; c <= m_nVariables - 2; c++) { /* -2 to leave room for d */
                for (int32_t d = c + 1; d <= m_nVariables - 1; d++) {

                    for (int32_t e = 1; e <= m_nVariables - 2; e++) {
                        for (int32_t f = e + 1; f <= m_nVariables - 1; f++) {

                            // Keep the function for later evaluation
                            Function_0_a_b_c_d_ef func(a, b, c, d, e, f, m_nVariables);
                            if (!func.isCanonicalForm()) {
                                continue;
                            }

                            h_funcArray[enqueued++] = func;

                            if (enqueued == GPUProps.actualConcurrentThreads) {
                                sendAndReport<Function_0_a_b_c_d_ef>(h_funcArray, d_funcArray, h_isMaxLength,
                                        d_isMaxLength, enqueued, m_maxPossibleLength);
                                enqueued = 0;
                            }

                        }
                    }
                }
            }
        }
    }

    if (enqueued != 0) {
        sendAndReport<Function_0_a_b_c_d_ef>(h_funcArray, d_funcArray, h_isMaxLength,
                d_isMaxLength, enqueued, m_maxPossibleLength);
    }

    hipFree(d_funcArray);
    hipFree(d_isMaxLength);
    delete [] h_isMaxLength;
    delete [] h_funcArray;
}






/***********************************************************************
 **************** For x0 + xa + xb + xc.xd.xe *******************
 ***********************************************************************/

    FuncGenerator_0_a_b_cde::FuncGenerator_0_a_b_cde(uint32_t nVariables)
: m_nVariables(nVariables), m_maxPossibleLength((1 << m_nVariables) - 1)
{}


FuncGenerator_0_a_b_cde::~FuncGenerator_0_a_b_cde()
{}


void FuncGenerator_0_a_b_cde::reportMaxFunctions()
{
    getGPUProperties();

    hipError_t ret;

    Function_0_a_b_cde *h_funcArray;
    Function_0_a_b_cde *d_funcArray;
    bool *d_isMaxLength;
    bool *h_isMaxLength;

    h_isMaxLength = new bool[GPUProps.actualConcurrentThreads];
    h_funcArray = new Function_0_a_b_cde[GPUProps.actualConcurrentThreads];

    ret = hipMalloc((void **) &d_isMaxLength, GPUProps.actualConcurrentThreads * sizeof(bool));
    if (ret != hipSuccess) {
        throw std::runtime_error("No more memory on device for bools");
    }

    ret = hipMalloc((void **) &d_funcArray, GPUProps.actualConcurrentThreads * sizeof(Function_0_a_b_cde));
    if (ret != hipSuccess) {
        throw std::runtime_error("No more memory on device for Function_0_a_b_cde");
    }

    uint32_t enqueued = 0;

    for (int32_t a = 1; a <= (m_nVariables + 1) / 2; a++) {
        for (int32_t b = a + 1; b <= m_nVariables - 1; b++) {


            for (int32_t c = 1; c <= m_nVariables - 3; c++) { /* -3 to leave room for d and e */
                for (int32_t d = c + 1; d <= m_nVariables - 2; d++) {
                    for (int32_t e = d + 1; e <= m_nVariables - 1; e++) {

                        // Keep the function for later evaluation
                        Function_0_a_b_cde func(a, b, c, d, e, m_nVariables);
                        if (!func.isCanonicalForm()) {
                            continue;
                        }

                        h_funcArray[enqueued++] = func;

                        if (enqueued == GPUProps.actualConcurrentThreads) {
                            sendAndReport<Function_0_a_b_cde>(h_funcArray, d_funcArray, h_isMaxLength,
                                    d_isMaxLength, enqueued, m_maxPossibleLength);
                            enqueued = 0;
                        }

                    }
                }
            }
        }
    }

    if (enqueued != 0) {
        sendAndReport<Function_0_a_b_cde>(h_funcArray, d_funcArray, h_isMaxLength,
                d_isMaxLength, enqueued, m_maxPossibleLength);
    }

    hipFree(d_funcArray);
    hipFree(d_isMaxLength);
    delete [] h_isMaxLength;
    delete [] h_funcArray;
}
#endif


int main(int argc, char *argv[])
{
    report_0_a_b_cd(atoi(argv[1]));
    return 0;
}
